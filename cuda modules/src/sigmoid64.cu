
#include <hip/hip_runtime.h>
#include <math.h>

#ifdef __cplusplus
extern "C" {
#endif


__global__ void sigmoid32(double* A, int size)
{
	int idx = threadIdx.x;
	if (idx >= size) {
		return;
	}
	A[idx] = 1 / (1 + pow(M_E, (double)(-1 * A[idx])));
}
	
#ifdef __cplusplus
}
#endif